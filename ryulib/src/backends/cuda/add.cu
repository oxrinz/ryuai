
#include <hip/hip_runtime.h>
#include <stdio.h> 

extern "C" __global__ void addKernel(int *a, int *b, int *c)
{
    *c = *a + *b;
}

extern "C" void launchAddKernel(int *a, int *b, int *c) 
{
    addKernel<<<1, 1>>>(a, b, c); 
}